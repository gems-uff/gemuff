#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include "K_IMUFF.h"

__global__ void diff_k(uchar4* src1data, uchar4* src2data, uchar4* delta, int totalElements)
{
        int idx = blockDim.x * blockIdx.x + threadIdx.x;

        if (idx < totalElements)
        {
                uchar4 src1 = src1data[idx];
                uchar4 src2 = src2data[idx];

                //if ((src1.x == src2.x) && (src1.y == src2.y) && (src1.z == src2.z))
                //	data[idx] = make_uchar4(0, 0, 0, 0);
                //else
                //	data[idx] = make_uchar4(255, 255, 255, 255);

                //data[idx] = make_int4(src1.x - src2.x, src1.y - src2.y,
                //	src1.z - src2.z, src1.w - src2.w);

                delta[idx] = make_uchar4(src1.x xor src2.x, src1.y xor src2.y, src1.z xor src2.z, src1.w xor src2.w);

        }
}

__global__ void fromRGBToYCbCr(uchar4* src1data, uchar4* out, int totalElements){

}


/*
 * Funcao global em GPU para aplicacao da diferenca em uma imagem
 * imagem = imagem_base + delta
 */
__global__ void patch_k(uchar4* src1data, uchar4* delta, uchar4* outimage, int totalElements)
{
        int idx = blockDim.x * blockIdx.x + threadIdx.x;

        if (idx < totalElements)
        {
                uchar4 src1 = src1data[idx];
                uchar4 src2 = delta[idx];

                //if ((src1.x == src2.x) && (src1.y == src2.y) && (src1.z == src2.z))
                //	data[idx] = make_uchar4(0, 0, 0, 0);
                //else
                //	data[idx] = make_uchar4(255, 255, 255, 255);

                //data[idx] = make_int4(src1.x - src2.x, src1.y - src2.y,
                //	src1.z - src2.z, src1.w - src2.w);

                outimage[idx] = make_uchar4(src1.x xor src2.x, src1.y xor src2.y, src1.z xor src2.z, src1.w xor src2.w);

        }
}


extern "C"
    void gIMUFFDiff(const uchar* src1data, const uchar* src2data, void* delta, int totalElements)
    {
        int numElements = totalElements;

        uchar4 *d_data1, *d_data2, *d_delta;
        checkCudaErrors(hipMalloc((void**)&d_data1, sizeof(uchar4) * numElements));
        checkCudaErrors(hipMalloc((void**)&d_data2, sizeof(uchar4) * numElements));
        checkCudaErrors(hipMalloc((void**)&d_delta, sizeof(uchar4) * numElements));

        checkCudaErrors(hipMemcpy((void*)d_data1, src1data,
                        sizeof(uchar4) * numElements, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy((void*)d_data2, src2data,
                        sizeof(uchar4) * numElements, hipMemcpyHostToDevice));

        // Processamento
        dim3 dimBlock(256, 1, 1);
        dim3 dimGrid((numElements/dimBlock.x) + 1, 1);
        diff_k<<<dimGrid, dimBlock>>>(d_data1, d_data2, d_delta, numElements);

        checkCudaErrors(hipMemcpy((void*)delta, d_delta,
                        sizeof(uchar4) * numElements, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_data1));
        checkCudaErrors(hipFree(d_data2));
        checkCudaErrors(hipFree(d_delta));
    }


extern "C"
    void gIMUFFPatch(const uchar* src1data, const uchar* delta, void* imgfinal, int totalElements)
    {

        int numElements = totalElements;

        uchar4 *d_data1,*d_delta, *d_data2;
        checkCudaErrors(hipMalloc((void**)&d_data1, sizeof(uchar4) * numElements));
        checkCudaErrors(hipMalloc((void**)&d_data2, sizeof(uchar4) * numElements));
        checkCudaErrors(hipMalloc((void**)&d_delta, sizeof(uchar4) * numElements));

        checkCudaErrors(hipMemcpy((void*)d_data1, src1data,
                        sizeof(uchar4) * numElements, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy((void*)d_delta, delta,
                        sizeof(uchar4) * numElements, hipMemcpyHostToDevice));

        // Processamento
        dim3 dimBlock(256, 1, 1);
        dim3 dimGrid((numElements/dimBlock.x) + 1, 1);
        patch_k<<<dimGrid, dimBlock>>>(d_data1, d_delta, d_data2, numElements);

        checkCudaErrors(hipMemcpy((void*)imgfinal, d_data2,
                        sizeof(uchar4) * numElements, hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(d_data1));
        checkCudaErrors(hipFree(d_data2));
        checkCudaErrors(hipFree(d_delta));
    }


extern "C"
    void gIMUFFMerge(const uchar* imgbase, const uchar* imgA, const uchar* imgB, void* outimage, int totalElements)
    {
        int numElements = totalElements;

        // Calcular delta da imagem base e B
        uchar4 *d_imgbase, *d_imgA, *d_delta;
        checkCudaErrors(hipMalloc((void**)&d_imgbase, sizeof(uchar4) * numElements));
        checkCudaErrors(hipMalloc((void**)&d_imgA, sizeof(uchar4) * numElements));
        checkCudaErrors(hipMalloc((void**)&d_delta, sizeof(uchar4) * numElements));

        checkCudaErrors(hipMemcpy((void*)d_imgbase, imgbase,
            sizeof(uchar4) * numElements, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy((void*)d_imgA, imgA,
            sizeof(uchar4) * numElements, hipMemcpyHostToDevice));

        // Processamento do delta
        dim3 dimBlock(256, 1, 1);
        dim3 dimGrid((numElements/dimBlock.x) + 1, 1);
        diff_k<<<dimGrid, dimBlock>>>(d_imgbase, d_imgA, d_delta, numElements);

        // Aplicar patch do delta na imagem B
        uchar4 *d_imgB, *d_outimage;
        checkCudaErrors(hipMalloc((void**)&d_imgB, sizeof(uchar4) * numElements));
        checkCudaErrors(hipMalloc((void**)&d_outimage, sizeof(uchar4) * numElements));
        checkCudaErrors(hipMemcpy((void*)d_imgB, imgB,
            sizeof(uchar4) * numElements, hipMemcpyHostToDevice));

        patch_k<<<dimGrid, dimBlock>>>(d_imgB, d_delta, d_outimage, numElements);

        checkCudaErrors(hipMemcpy((void*)outimage, d_outimage,
            sizeof(uchar4) * numElements, hipMemcpyDeviceToHost));

         checkCudaErrors(hipFree(d_imgbase));
         checkCudaErrors(hipFree(d_imgA));
         checkCudaErrors(hipFree(d_imgB));
         checkCudaErrors(hipFree(d_delta));
         checkCudaErrors(hipFree(d_outimage));
    }
